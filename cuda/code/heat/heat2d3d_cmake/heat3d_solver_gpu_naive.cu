#include "hip/hip_runtime.h"
/**
 * \file heat3d_solver_gpu_naive.cu
 * \brief Solve 3D heat equation (finite difference method). GPU version (naive).
 *
 * We solve the 3D Heat equation \f$\partial_t \phi = \alpha \left[
 * \partial^2_x \phi + \partial^2_y \phi + \partial^2_z \ phi \right] \f$, \f$ 0 \leq x
 * \leq L_x \f$, \f$ 0 \leq y \leq L_y \f$, \f$ 0 \leq t\f$.\\
 *
 * Method : Finite Difference, FTCS scheme
 *
 * GPU Features: use only global memory
 *
 * boundary condition : Dirichlet
 *
 * GPU version : naive
 *
 * \date 27-dec-2009.
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <sys/time.h> // for gettimeofday
#include <assert.h>

#ifdef _OPENMP
#include <omp.h>
#endif
#include "openmp_utils.h"

// includes, project
//#include <helper_functions.h>
#include "cuda_error.h"
#include "CudaTimer.h"
#include "Timer.h"

// parameters and real_t typedef
#include "param.h"

// for output results
#include "output.h"

// GPU solver
#include "heat3d_kernel_gpu_naive.cu"

// CPU solver
#include "heat_kernel_cpu.h"

// initial conditions
#include "misc.h"

// cuda helper
#include "cuda_helper.h"

/////////////////////////////////////////////////////////////////////////
// declaration, forward
int runTest( int argc, char** argv);

/////////////////////////////////////////////////////////////////////////
// Program main
/////////////////////////////////////////////////////////////////////////
int
main(int argc, char** argv) 
{
  int status = runTest(argc, argv);

  CUDA_API_CHECK ( hipDeviceSynchronize() );
  CUDA_API_CHECK ( hipDeviceReset() );

  return status;
}

/////////////////////////////////////////////////////////////////////////
//! Run solver on GPU
/////////////////////////////////////////////////////////////////////////
int
runTest(int argc, char** argv) 
{
  int devID;
  hipDeviceProp_t deviceProps;
  
  devID = findCudaDevice(argc, (const char **)argv);
  
  // get number of SMs on this GPU
  CUDA_API_CHECK( hipGetDeviceProperties(&deviceProps, devID) );
  printf("CUDA device [%s] has %d Multi-Processors\n", deviceProps.name, deviceProps.multiProcessorCount);

  /*
   * read and print parameters
   */
  // default parameter file
  std::string paramFile("heatEqSolver.par");

  // if argv[1] exists use it as a parameter file
  if (argc>1) {
    printf("trying to read parameters from file %s ...\n",argv[1]);
    paramFile = std::string(argv[1]);
  }

  // read parameter file
  readParamFile(paramFile);

  if (NZ<=1) {
    printf("NZ should be larger than 1 in the 3D version\n");
    hipDeviceReset();
  }

  // print parameters on screen
  printParameters("HEAT 3D - GPU (NAIVE)");

  CudaTimer gpuTimer;

  unsigned int mem_size = sizeof(real_t)*NX*NY*NZ;

  // allocate host memory
  real_t* data1 = (real_t*) malloc( mem_size);
  real_t* data2 = (real_t*) malloc( mem_size);
  
  ///////////////////////////////////////////////////
  // compute GPU solution to 3D heat equation
  ///////////////////////////////////////////////////
  
  // inital condition
  initCondition3D (data1);

  // allocate device memory
  real_t* d_data1;
  real_t* d_data2;

  // device memory allocation (using hipMalloc)
  CUDA_API_CHECK( hipMalloc( (void**) &d_data1, mem_size));
  CUDA_API_CHECK( hipMalloc( (void**) &d_data2, mem_size));

  // copy host memory to device
  CUDA_API_CHECK( hipMemcpy( d_data1, data1, mem_size,
                              hipMemcpyHostToDevice) );
  CUDA_API_CHECK( hipMemcpy( d_data2, data1, mem_size,
                              hipMemcpyHostToDevice) );
   
  // setup execution parameters for cuda kernel
  // grid dimension for naive kernel
  unsigned int threadsPerBlockX=32;
  unsigned int threadsPerBlockY=10;
  dim3  threads(threadsPerBlockX,threadsPerBlockY);
  dim3  grid( (NX+threads.x-1)/threads.x, (NY+threads.y-1)/threads.y );
    
  printf("grid  size : %u %u %u\n",grid.x,grid.y,grid.z);
  printf("block size : %u %u %u\n",threads.x,threads.y,threads.z);

  // start timer
  gpuTimer.start();

  // time loop executing naive kernel
  int iTime   =  0;
  int iOutput = -1;
  for (iTime=0; iTime*DT<TMAX; ++iTime) {

    real_t* data_in  = iTime%2 == 0 ? d_data1 : d_data2;
    real_t* data_out = iTime%2 == 0 ? d_data2 : d_data1;

    if (useOrder2) { // use the 2nd order accurate scheme
      
      heat3d_ftcs_naive_order2_kernel<<< grid, threads >>>( data_in, data_out,
							    NX, NY, NZ,
							    o2.R, o2.R3);
      CUDA_KERNEL_CHECK("Kernel execution failed");
      
    } else if (useOrder2b) {
      
      heat3d_ftcs_naive_order2b_kernel<<< grid, threads >>>( data_in, data_out,
							     NX, NY, NZ,
							     o2.R, o2.R3b);
      CUDA_KERNEL_CHECK("Kernel execution failed");
      
    } else { // use the 4th order accurate scheme

      heat3d_ftcs_naive_order4_kernel<<< grid, threads >>>( data_in, data_out,
							    NX, NY, NZ,
							    o4.S, o4.S3);
      CUDA_KERNEL_CHECK("Kernel execution failed");
      
    }

    /* save output (just for cross-checking, do not save when
       measuring computing time */
    if (ENABLE_GPU_SAVE) {

      if (iTime%T_OUTPUT == 0) {
	iOutput++;
	CUDA_API_CHECK( hipMemcpy( data1, data_out, NX*NY*NZ*sizeof( real_t),
                                    hipMemcpyDeviceToHost) );      
      }
      // VTK output
      if (SAVE_VTK and iTime%T_OUTPUT == 0)
	save_vtk(data1, "heat3d_gpu_naive_",iOutput);

      // HDF5 output
      if (SAVE_HDF5 and iTime%T_OUTPUT == 0)
	save_hdf5(data1, "heat3d_gpu_naive_",iOutput);

    }

  } // end for loop
  
  // stop timer
  gpuTimer.stop();

  real_t gpu_time = gpuTimer.elapsed();
  printf( "GPU Processing time: %f (s)\n", gpu_time);
  
  // copy result from device to host
  real_t *resGPU = (real_t*) malloc( mem_size);
  CUDA_API_CHECK( hipMemcpy( resGPU, d_data1, mem_size,
			      hipMemcpyDeviceToHost) );
    
  if (SAVE_HDF5)
    write_xdmf_wrapper("heat3d_gpu_naive",N_ITER,T_OUTPUT);

  ////////////////////////////////////////////////////////
  // compute reference (CPU) solution to 3D heat equation
  // for performance comparison
  ////////////////////////////////////////////////////////
  printf("compute CPU reference solution\n");
  initCondition3D (data1);
  initCondition3D (data2);

  print_openmp_status();

  Timer cpuTimer;
  cpuTimer.start();
  
  // time loop
  iTime=0;
  for (iTime=0; iTime*DT<TMAX; ++iTime) {
    
    real_t* data_in  = iTime%2 == 0 ? data1 : data2;
    real_t* data_out = iTime%2 == 0 ? data2 : data1;

    if (useOrder2) {
    
      heat3d_ftcs_cpu_order2( data_in, data_out );
      
    } else if (useOrder2b) {
      
      //heat3d_ftcs_cpu_order2b( data_in, data_out );
        
    } else {

      heat3d_ftcs_cpu_order4( data_in, data_out );
      
    }
  }

  // stop timer
  cpuTimer.stop();
  real_t cpu_time = cpuTimer.elapsed();
  
  printf( "CPU Processing time: %g (s)\n", cpu_time);
  printf( "Speedup GPU/CPU : %f\n",cpu_time/gpu_time);

  printf("...comparing the results\n");
  double sum = 0, delta = 0;
  for(unsigned i = 0; i < NX*NY*NZ; i++){
    delta += (resGPU[i] - data1[i]) * (resGPU[i] - data1[i]);
    sum   += data1[i] * data1[i];
  }
  double L2norm = sqrt(delta / sum);
  printf("iteration %d relative L2 norm: %E\n", iTime, L2norm);

  // cuda device prop (to compute max bandwidth
  hipDeviceProp_t deviceProp;
  int deviceId;
  hipGetDevice(&deviceId);
  hipGetDeviceProperties(&deviceProp, deviceId);
  //printf("GPU mem clock rate in kHz %d\n",deviceProp.memoryClockRate);
  //printf("GPU mem bus width %d\n",deviceProp.memoryBusWidth);
  real_t gpuMaxBW = 1e-9*deviceProp.memoryClockRate*1000*(deviceProp.memoryBusWidth/8)*2;
  
  // bandwidth
  double totalBytes;
  if (useOrder2)
    totalBytes = NX*NY*NZ*sizeof(real_t) * (1*6+1 + 1) * iTime;
  else if (useOrder2b)
    totalBytes = NX*NY*NZ*sizeof(real_t) * (3*3*3 + 1) * iTime;
  else
    totalBytes = NX*NY*NZ*sizeof(real_t) * (2*6+1 + 1) * iTime;
  printf("CPU Bandwidth %f GBytes/s\n", totalBytes/cpu_time*1e-9);
  printf("GPU Bandwidth %f GBytes/s out of %f (%6.2f %%)\n", totalBytes/gpu_time*1e-9,gpuMaxBW,totalBytes/gpu_time*1e-9/gpuMaxBW*100);

  // cleanup memory
  free(data1);
  free(data2);
  free(resGPU);
  
  CUDA_API_CHECK( hipFree(d_data1) );
  CUDA_API_CHECK( hipFree(d_data2) );
  
  return EXIT_SUCCESS;
}
