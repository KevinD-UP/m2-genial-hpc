#include "hip/hip_runtime.h"
/**
 * \file heat2d_solver_gpu_shmem2.cu
 * \brief Solve 2D heat equation (finite difference method). GPU version (shmem2).
 *
 * We solve the 2D Heat equation \f$\partial_t \phi = \alpha \left[
 * \partial^2_x \phi + \partial^2_y \phi \right] \f$, \f$ 0 \leq x
 * \leq L_x \f$, \f$ 0 \leq y \leq L_y \f$, \f$ 0 \leq t\f$.\\
 *
 * Method : Finite Difference, FTCS scheme
 *
 * boundary condition : Dirichlet
 *
 * GPU version : shared memory 2
 *
 * \author Pierre Kestener.
 * \date 17-dec-2009.
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <sys/time.h> // for gettimeofday

#ifdef _OPENMP
#include <omp.h>
#endif
#include "openmp_utils.h"

// includes, project
//#include <helper_functions.h>
#include "cuda_error.h"
#include "CudaTimer.h"
#include "Timer.h"

// parameters and real_t typedef
#include "param.h"

// for output results
#include "output.h"

// GPU solver
#include "heat2d_kernel_gpu_shmem2.cu"

// CPU solver
#include "heat_kernel_cpu.h"

// initial conditions
#include "misc.h"

// cuda helper
#include "cuda_helper.h"

/////////////////////////////////////////////////////////////////////////
// declaration, forward
int runTest( int argc, char** argv);

/////////////////////////////////////////////////////////////////////////
// Program main
/////////////////////////////////////////////////////////////////////////
int
main(int argc, char** argv) 
{
  int status = runTest(argc, argv);

  CUDA_API_CHECK ( hipDeviceSynchronize() );
  CUDA_API_CHECK ( hipDeviceReset() );

  return status;
}

/////////////////////////////////////////////////////////////////////////
//! Run solver on GPU
/////////////////////////////////////////////////////////////////////////
int
runTest(int argc, char** argv) 
{
  int devID;
  hipDeviceProp_t deviceProps;
  
  devID = findCudaDevice(argc, (const char **)argv);
  
  // get number of SMs on this GPU
  CUDA_API_CHECK( hipGetDeviceProperties(&deviceProps, devID) );
  printf("CUDA device [%s] has %d Multi-Processors\n", deviceProps.name, deviceProps.multiProcessorCount);

  /*
   * read and print parameters
   */
  // default parameter file
  std::string paramFile("heatEqSolver.par");

  // if argv[1] exists use it as a parameter file
  if (argc>1) {
    printf("trying to read parameters from file %s ...\n",argv[1]);
    paramFile = std::string(argv[1]);
  }

  // read parameter file
  readParamFile(paramFile);

  // print parameters on screen
  printParameters("HEAT 2D - GPU (SHMEM2)");

  CudaTimer gpuTimer;

  unsigned int mem_size = sizeof(real_t)*NX*NY;

  // allocate host memory
  real_t* data1 = (real_t*) malloc( mem_size);
  real_t* data2 = (real_t*) malloc( mem_size);
  
  ///////////////////////////////////////////////////
  // compute GPU solution to 2D heat equation
  ///////////////////////////////////////////////////
  
  // inital condition
  initCondition2D (data1);

  // allocate device memory
  real_t* d_data1;
  real_t* d_data2;

  // device memory allocation (using hipMalloc)
  CUDA_API_CHECK( hipMalloc( (void**) &d_data1, mem_size));
  CUDA_API_CHECK( hipMalloc( (void**) &d_data2, mem_size));

  // copy host memory to device
  CUDA_API_CHECK( hipMemcpy( d_data1, data1, mem_size,
                              hipMemcpyHostToDevice) );
  CUDA_API_CHECK( hipMemcpy( d_data2, data1, mem_size,
                              hipMemcpyHostToDevice) );
    
  // setup execution parameters for cuda kernel
  // grid dimension for sharedmem kernel
  dim3  threads;
  dim3  grid;
  if (useOrder2 or useOrder2b) {
    threads.x = BLOCK_DIMX;
    threads.y = BLOCK_DIMY;
    grid.x    = (NX+BLOCK_DIMX-1)/BLOCK_DIMX;
    grid.y    = (NY+BLOCK_DIMY-1)/BLOCK_DIMY;
  } else { // 4th order
    threads.x = BLOCK_DIMX2;
    threads.y = BLOCK_DIMY2;    
    grid.x    = (NX+BLOCK_DIMX2-1)/BLOCK_DIMX2;
    grid.y    = (NY+BLOCK_DIMY2-1)/BLOCK_DIMY2;
  }

  printf("grid  size : %u %u\n",grid.x,grid.y);
  printf("block size : %u %u\n",threads.x,threads.y);
  
  // copy scheme parameters to device constant memory
  hipMemcpyToSymbol(HIP_SYMBOL(::o2Gpu), &o2, sizeof(struct SecondOrderParam),0,hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(::o4Gpu), &o4, sizeof(struct FourthOrderParam),0,hipMemcpyHostToDevice);

  // start timer
  gpuTimer.start();

  // time loop executing shared mem kernel
  int iTime   =  0;
  int iOutput = -1;
  for (iTime=0; iTime*DT<TMAX; ++iTime) {

    real_t* data_in  = iTime%2 == 0 ? d_data1 : d_data2;
    real_t* data_out = iTime%2 == 0 ? d_data2 : d_data1;

    if (useOrder2) { // use the 2nd order accurate scheme
      
      heat2d_ftcs_sharedmem2_order2_kernel<<< grid, threads >>>( data_in, data_out,
								 NX, NY);
      CUDA_KERNEL_CHECK("Kernel execution failed");
      
    } else { // use the 4th order accurate scheme
      
      heat2d_ftcs_sharedmem2_order4_kernel<<< grid, threads >>>( data_in, data_out, 
								 NX, NY);
      CUDA_KERNEL_CHECK("Kernel execution failed");
      
    }

    /* save output (just for cross-checking, do not save when
       measuring computing time */
    if (ENABLE_GPU_SAVE) {

      if (iTime%T_OUTPUT == 0) {
	iOutput++;
	CUDA_API_CHECK( hipMemcpy( data1, data_out, mem_size,
				    hipMemcpyDeviceToHost) );
      }
      // PGM output
      if (SAVE_PGM and iTime%T_OUTPUT == 0)
	save_pgm(data1, "heat2d_gpu_shmem2_",iOutput,NX,NY);
      
      // MathGL save (3D view)
      if (SAVE_MGL and iTime%T_OUTPUT == 0)
	save_mgl(data1, "heat2d_gpu_shmem2_",iOutput,NX,NY);

      // VTK output
      if (SAVE_VTK and iTime%T_OUTPUT == 0)
	save_vtk(data1, "heat2d_gpu_shmem2_",iOutput);

      // HDF5 output
      if (SAVE_HDF5 and iTime%T_OUTPUT == 0)
	save_hdf5(data1, "heat2d_gpu_shmem2_",iOutput);

    }

  } // end for loop
  
  // stop timer
  gpuTimer.stop();

  real_t gpu_time = gpuTimer.elapsed();
  printf( "GPU Processing time: %f (s)\n", gpu_time);
  
  // copy result from device to host
  real_t *resGPU = (real_t*) malloc( mem_size);
  CUDA_API_CHECK( hipMemcpy( resGPU, d_data1, mem_size,
                              hipMemcpyDeviceToHost) );
    
  if (SAVE_HDF5)
    write_xdmf_wrapper("heat2d_gpu_shmem2",N_ITER,T_OUTPUT);

  ////////////////////////////////////////////////////////
  // compute reference (CPU) solution to 2D heat equation
  // for performance comparison
  ////////////////////////////////////////////////////////
  printf("compute CPU reference solution\n");
  initCondition2D (data1);
  initCondition2D (data2);

  print_openmp_status();

  Timer cpuTimer;
  cpuTimer.start();
  
  // time loop
  iTime=0;
  for (iTime=0; iTime*DT<TMAX; ++iTime) {
    
    real_t* data_in  = iTime%2 == 0 ? data1 : data2;
    real_t* data_out = iTime%2 == 0 ? data2 : data1;

    if (useOrder2) {
    
      heat2d_ftcs_cpu_order2( data_in, data_out );
      
    } else if (useOrder2b) {

      heat2d_ftcs_cpu_order2b( data_in, data_out );
      
    } else {

      heat2d_ftcs_cpu_order4( data_in, data_out );
      
    }
  }

  // stop timer
  cpuTimer.stop();
  real_t cpu_time = cpuTimer.elapsed();
  
  printf( "CPU Processing time: %g (s)\n", cpu_time);
  printf( "Speedup GPU/CPU : %f\n",cpu_time/gpu_time);

  printf("...comparing the results\n");
  double sum = 0, delta = 0;
  for(unsigned i = 0; i < NX*NY; i++){
    delta += (resGPU[i] - data1[i]) * (resGPU[i] - data1[i]);
    sum   += data1[i] * data1[i];
    if (abs(resGPU[i] - data1[i]) > 1e-5)
      printf("i j : %d %d\n",i-(i/NX)*NX,i/NX);
  }
  double L2norm = sqrt(delta / sum);
  printf("iteration %d relative L2 norm: %.10g\n", iTime, L2norm);

  // cuda device prop (to compute max bandwidth
  hipDeviceProp_t deviceProp;
  int deviceId;
  hipGetDevice(&deviceId);
  hipGetDeviceProperties(&deviceProp, deviceId);
  //printf("GPU mem clock rate in kHz %d\n",deviceProp.memoryClockRate);
  //printf("GPU mem bus width %d\n",deviceProp.memoryBusWidth);
  real_t gpuMaxBW = 1e-9*deviceProp.memoryClockRate*1000*(deviceProp.memoryBusWidth/8)*2;
  
  // bandwidth
  double totalBytes;
  if (useOrder2)
    totalBytes = NX*NY*sizeof(real_t) * (5 + 1) * iTime;
  else if (useOrder2b)
    totalBytes = NX*NY*sizeof(real_t) * (3*3 + 1) * iTime;
  else
    totalBytes = NX*NY*sizeof(real_t) * (9 + 1) * iTime;
  printf("CPU Bandwidth %f GBytes/s\n", totalBytes/cpu_time*1e-9);
  printf("GPU Bandwidth %f GBytes/s out of %f (%6.2f %%)\n", totalBytes/gpu_time*1e-9,gpuMaxBW,totalBytes/gpu_time*1e-9/gpuMaxBW*100);

  // cleanup memory
  free(data1);
  free(data2);
  free(resGPU);
  
  CUDA_API_CHECK( hipFree(d_data1) );
  CUDA_API_CHECK( hipFree(d_data2) );
  
  return EXIT_SUCCESS;
}
