#include "hip/hip_runtime.h"
/**
 * \file heat3d_solver_gpu_shmem2.cu
 * \brief Solve 3D heat equation (finite difference method). GPU version (shmem2).
 *
 * We solve the 3D Heat equation \f$\partial_t \phi = \alpha \left[
 * \partial^2_x \phi + \partial^2_y \phi + \partial^2_z \ phi \right] \f$, \f$ 0 \leq x
 * \leq L_x \f$, \f$ 0 \leq y \leq L_y \f$, \f$ 0 \leq t\f$.\\
 *
 * Method : Finite Difference, FTCS scheme
 *
 * boundary condition : Dirichlet
 *
 * GPU version : shared memory 2
 *
 * \date 27-dec-2009.
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <sys/time.h> // for gettimeofday
#include <assert.h>

#ifdef _OPENMP
#include <omp.h>
#endif
#include "openmp_utils.h"

// includes, project
//#include <helper_functions.h>
#include "cuda_error.h"
#include "CudaTimer.h"
#include "Timer.h"

// parameters and real_t typedef
#include "param.h"

// for output results
#include "output.h"

// GPU solver
#include "heat3d_kernel_gpu_shmem2.cu"

// CPU solver
#include "heat_kernel_cpu.h"

// initial conditions
#include "misc.h"

// cuda helper
#include "cuda_helper.h"

/////////////////////////////////////////////////////////////////////////
// declaration, forward
int runTest( int argc, char** argv);

/////////////////////////////////////////////////////////////////////////
// Program main
/////////////////////////////////////////////////////////////////////////
int
main(int argc, char** argv) 
{
  int status = runTest(argc, argv);

  CUDA_API_CHECK ( hipDeviceSynchronize() );
  CUDA_API_CHECK ( hipDeviceReset() );

  return status;
}

/////////////////////////////////////////////////////////////////////////
//! Run solver on GPU
/////////////////////////////////////////////////////////////////////////
int
runTest(int argc, char** argv) 
{
  int devID;
  hipDeviceProp_t deviceProps;
  
  devID = findCudaDevice(argc, (const char **)argv);
  
  // get number of SMs on this GPU
  CUDA_API_CHECK( hipGetDeviceProperties(&deviceProps, devID) );
  printf("CUDA device [%s] has %d Multi-Processors\n", deviceProps.name, deviceProps.multiProcessorCount);

  /*
   * read and print parameters
   */
  // default parameter file
  std::string paramFile("heatEqSolver.par");

  // if argv[1] exists use it as a parameter file
  if (argc>1) {
    printf("trying to read parameters from file %s ...\n",argv[1]);
    paramFile = std::string(argv[1]);
  }

  // read parameter file
  readParamFile(paramFile);

  if (NZ<=1) {
    printf("NZ should be larger than 1 in the 3D version\n");
    hipDeviceReset();
  }

  // print parameters on screen
  printParameters("HEAT 3D - GPU (SHMEM2)");

  CudaTimer gpuTimer;

  unsigned int mem_size = sizeof(real_t)*NX*NY*NZ;

  // allocate host memory
  real_t* data1 = (real_t*) malloc( mem_size);
  real_t* data2 = (real_t*) malloc( mem_size);
  
  ///////////////////////////////////////////////////
  // compute GPU solution to 3D heat equation
  ///////////////////////////////////////////////////
  
  // inital condition
  initCondition3D (data1);

  // allocate device memory
  real_t* d_data1;
  real_t* d_data2;

  // device memory allocation (using hipMalloc)
  CUDA_API_CHECK( hipMalloc( (void**) &d_data1, mem_size));
  CUDA_API_CHECK( hipMalloc( (void**) &d_data2, mem_size));

  // copy host memory to device
  CUDA_API_CHECK( hipMemcpy( d_data1, data1, mem_size,
                              hipMemcpyHostToDevice) );
  CUDA_API_CHECK( hipMemcpy( d_data2, data1, mem_size,
                              hipMemcpyHostToDevice) );
    
  // setup execution parameters for cuda kernel
  // grid dimension for sharedmem2 kernel
  dim3  threads;
  dim3  grid;
  if (useOrder2) {
    threads.x = BLOCK_DIMX;
    threads.y = BLOCK_DIMY;
    grid.x    = (NX+BLOCK_DIMX-1)/BLOCK_DIMX;
    grid.y    = (NY+BLOCK_DIMY-1)/BLOCK_DIMY;
  } else { // 4th order
    threads.x = BLOCK_DIMX2;
    threads.y = BLOCK_DIMY2;
    grid.x    = (NX+BLOCK_DIMX2-1)/BLOCK_DIMX2;
    grid.y    = (NY+BLOCK_DIMY2-1)/BLOCK_DIMY2;
  }

  printf("grid  size : %u %u\n",grid.x,grid.y);
  printf("block size : %u %u\n",threads.x,threads.y);
  
  // copy scheme parameters to device constant memory
  hipMemcpyToSymbol(HIP_SYMBOL(::o2Gpu), &o2, sizeof(struct SecondOrderParam),0,hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(::o4Gpu), &o4, sizeof(struct FourthOrderParam),0,hipMemcpyHostToDevice);

  // start timer
  gpuTimer.start();

  // time loop executing sharedmem2 kernel
  int iTime   =  0;
  int iOutput = -1;
  for (iTime=0; iTime*DT<TMAX; ++iTime) {

    real_t* data_in  = iTime%2 == 0 ? d_data1 : d_data2;
    real_t* data_out = iTime%2 == 0 ? d_data2 : d_data1;

    if (useOrder2) { // use the 2nd order accurate scheme
      
      heat3d_ftcs_sharedmem2_order2_kernel<<< grid, threads >>>( data_in,
								 data_out,
								 NX, NY, NZ);
      CUDA_KERNEL_CHECK("Kernel execution failed");
          
    } else { // use the 4th order accurate scheme

      heat3d_ftcs_sharedmem2_order4_kernel<<< grid, threads >>>( data_in, 
								 data_out,
								 NX, NY, NZ);
      CUDA_KERNEL_CHECK("Kernel execution failed");
      
    }

    /* save output (just for cross-checking, do not save when
       measuring computing time */
    if (ENABLE_GPU_SAVE) {

      if (iTime%T_OUTPUT == 0) {
	iOutput++;
	CUDA_API_CHECK( hipMemcpy( data1, data_out, mem_size,
                                    hipMemcpyDeviceToHost) );
      }
      // VTK output
      if (SAVE_VTK and iTime%T_OUTPUT == 0)
	save_vtk(data1, "heat3d_gpu_shmem2_",iOutput);

      // HDF5 output
      if (SAVE_HDF5 and iTime%T_OUTPUT == 0)
	save_hdf5(data1, "heat3d_gpu_shmem2_",iOutput);

    }

  } // end for loop
  
  // stop timer
  gpuTimer.stop();

  real_t gpu_time = gpuTimer.elapsed();
  printf( "GPU Processing time: %f (s)\n", gpu_time);
  
  // copy result from device to host
  real_t *resGPU = (real_t*) malloc( mem_size);
  CUDA_API_CHECK( hipMemcpy( resGPU, d_data1, mem_size,
			      hipMemcpyDeviceToHost) );
    
  if (SAVE_HDF5)
    write_xdmf_wrapper("heat3d_gpu_shmem2",N_ITER,T_OUTPUT);

  ////////////////////////////////////////////////////////
  // compute reference (CPU) solution to 3D heat equation
  // for performance comparison
  ////////////////////////////////////////////////////////
  printf("compute CPU reference solution\n");
  initCondition3D (data1);
  initCondition3D (data2);

  print_openmp_status();

  Timer cpuTimer;
  cpuTimer.start();
  
  // time loop
  iTime=0;
  for (iTime=0; iTime*DT<TMAX; ++iTime) {
    
    real_t* data_in  = iTime%2 == 0 ? data1 : data2;
    real_t* data_out = iTime%2 == 0 ? data2 : data1;

    if (useOrder2) {
    
      heat3d_ftcs_cpu_order2( data_in, data_out );
      
    } else {

      heat3d_ftcs_cpu_order4( data_in, data_out );
      
    }
  }

  // stop timer
  cpuTimer.stop();
  real_t cpu_time = cpuTimer.elapsed();
  
  printf( "CPU Processing time: %g (s)\n", cpu_time);
  printf( "Speedup GPU/CPU : %f\n",cpu_time/gpu_time);

  printf("...comparing the results\n");
  double sum = 0, delta = 0;
  for(unsigned index = 0; index < NX*NY*NZ; index++){
    delta += (resGPU[index] - data1[index]) * (resGPU[index] - data1[index]);
    sum   += data1[index] * data1[index];
    if (abs(resGPU[index] - data1[index]) > 1e-4) {
      int i,j,k;
      k = index/(NX*NY);
      j = (index-k*NX*NY)/NX;
      i = index-k*NX*NY-j*NX;
      printf("i j k: %d %d %d\n",i,j,k);
    }
  }
  double L2norm = sqrt(delta / sum);
  printf("iteration %d relative L2 norm: %.10g\n", iTime, L2norm);

  // cuda device prop (to compute max bandwidth
  hipDeviceProp_t deviceProp;
  int deviceId;
  hipGetDevice(&deviceId);
  hipGetDeviceProperties(&deviceProp, deviceId);
  //printf("GPU mem clock rate in kHz %d\n",deviceProp.memoryClockRate);
  //printf("GPU mem bus width %d\n",deviceProp.memoryBusWidth);
  real_t gpuMaxBW = 1e-9*deviceProp.memoryClockRate*1000*(deviceProp.memoryBusWidth/8)*2;
  
  // bandwidth
  double totalBytes;
  if (useOrder2)
    totalBytes = NX*NY*NZ*sizeof(real_t) * (1*6+1 + 1) * iTime;
  else if (useOrder2b)
    totalBytes = NX*NY*NZ*sizeof(real_t) * (3*3*3 + 1) * iTime;
  else
    totalBytes = NX*NY*NZ*sizeof(real_t) * (2*6+1 + 1) * iTime;
  printf("CPU Bandwidth %f GBytes/s\n", totalBytes/cpu_time*1e-9);
  printf("GPU Bandwidth %f GBytes/s out of %f (%6.2f %%)\n", totalBytes/gpu_time*1e-9,gpuMaxBW,totalBytes/gpu_time*1e-9/gpuMaxBW*100);

  // cleanup memory
  free(data1);
  free(data2);
  free(resGPU);
  
  CUDA_API_CHECK( hipFree(d_data1) );
  CUDA_API_CHECK( hipFree(d_data2) );
  
  return EXIT_SUCCESS;
}
