
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

struct Array {
  int data[5];
};


__global__ void print_from_gpu(Array a) {
  printf("Hello World from GPU thread %d, block %d ! data = %d %d %d %d %d\n",
         threadIdx.x, blockIdx.x,
         a.data[0], a.data[1], a.data[2], a.data[3], a.data[4]);
}

int main(int argc, char* argv[]) 
{
  printf("Hello from CPU !\n");

  Array a = {0,1,2,3,4};

  print_from_gpu<<<1,1>>>(a);

  hipDeviceSynchronize();
  return EXIT_SUCCESS;
}
