/*
 * How to build:
 *
 * nvcc -arch=sm_80 -o helloworld_block helloworld_block.cu
 *
 * Note that you need to adjust the architecture version to your current GPU hardware.
 * Hardware version can be probed with e.g. deviceQuery example (from Nvidia SDK samples).
 *
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void hello()
{
  printf("I'm a thread (%d,%d) in block (%d,%d)\n", 
         threadIdx.x, threadIdx.y,
         blockIdx.x, blockIdx.y);
}


int main(int argc,char **argv)
{

  // default values for 
  // - gridSize :  number of blocks
  // - blockSize : number of threads per block
  unsigned int gridSize  = argc > 1 ? atoi(argv[1]) : 2;
  unsigned int blockSize = argc > 2 ? atoi(argv[2]) : 2;

  dim3 gridSize_2d  {gridSize,  gridSize};
  dim3 blockSize_2d {blockSize, blockSize};

  // launch the kernel
  hello<<<gridSize_2d, blockSize_2d>>>();
  
  // force the printf()s to flush
  hipDeviceSynchronize();
  
  printf("That's all!\n");
  
  return EXIT_SUCCESS;
}
